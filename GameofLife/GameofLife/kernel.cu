#include "hip/hip_runtime.h"
﻿// Include necessary libraries
#include <SDL.h>              // SDL for graphics
#include <hip/hip_runtime.h>     // CUDA runtime
#include <iostream>           // For standard I/O operations
#include <random>             // For random number generation
#include ""  // For CUDA kernel launch parameters

// Define constants for the game grid and window
const int GRID_WIDTH = 100;    // Width of the game grid
const int GRID_HEIGHT = 80;   // Height of the game grid
const int CELL_SIZE = 10;     // Size of each cell in pixels (smaller size now)
const int WINDOW_WIDTH = GRID_WIDTH * CELL_SIZE;   // Window width
const int WINDOW_HEIGHT = GRID_HEIGHT * CELL_SIZE; // Window height
const int BLOCK_SIZE = 2;    // CUDA block size
const int FRAME_DELAY = 9;  // Delay between frames in milliseconds to control the rendering speed
const int UPDATE_FREQUENCY = 6;  // Update grid every 5 frames

// CUDA kernel for updating the Game of Life grid
__global__ void updateGrid(unsigned char* d_grid, unsigned char* d_newGrid, int width, int height) { //This is the CUDA kernel function that runs on the GPU. It updates the state of the game grid.
    int x = blockIdx.x * blockDim.x + threadIdx.x; //Computes the x-coordinate for the cell that this thread is responsible for. Each thread in a CUDA block computes one cell.
    int y = blockIdx.y * blockDim.y + threadIdx.y; //Computes the y-coordinate similarly.

    if (x < width && y < height) {
        int idx = y * width + x; //Converts the (x, y) coordinates to a 1D index, which is used to access the 1D array representing the 2D grid.
        int count = 0;

        // Check all 8 neighboring cells
        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                if (dx == 0 && dy == 0) continue;
                int nx = (x + dx + width) % width;
                int ny = (y + dy + height) % height;
                count += d_grid[ny * width + nx];
            } //These loops check the eight neighbors around the current cell (ignoring the cell itself).
        }

        // Apply Game of Life rules
        d_newGrid[idx] = (count == 3 || (count == 2 && d_grid[idx])) ? 1 : 0; //If the cell is alive and has exactly 2 or 3 neighbors, it stays alive. If the cell is dead and has exactly 3 neighbors, it becomes alive. Otherwise, it stays dead.


    }
}

// Initialize the grid with a random pattern
void initializeGrid(unsigned char* grid, int width, int height) {
    std::random_device rd;  // Obtain a random number from hardware
    std::mt19937 gen(rd()); // Seed the generator. 
    std::uniform_int_distribution<> dis(0, 1);  // Define the range

    // Set each cell to either 0 or 1 randomly
    for (int i = 0; i < width * height; i++) {
        grid[i] = dis(gen); // iterates over the entire grid and assigns each cell a random value (0 or 1).

    }
}

int main(int argc, char* argv[]) {
    // Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0) { //Initializes SDL for video rendering.
        std::cerr << "SDL could not initialize! SDL_Error: " << SDL_GetError() << std::endl;
        return -1;
    }

    // Create window
    SDL_Window* window = SDL_CreateWindow("Conway's Game of Life", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, WINDOW_WIDTH, WINDOW_HEIGHT, SDL_WINDOW_SHOWN); //Creates the SDL window with the dimensions
    if (window == nullptr) {
        std::cerr << "Window could not be created! SDL_Error: " << SDL_GetError() << std::endl;
        return -1;
    }

    // Create renderer
    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED); //Creates the SDL renderer, which is used to draw the grid.
    if (renderer == nullptr) {
        std::cerr << "Renderer could not be created! SDL_Error: " << SDL_GetError() << std::endl;
        return -1;
    }

    // Allocate memory for the grid on the host
    unsigned char* h_grid = new unsigned char[GRID_WIDTH * GRID_HEIGHT]; //Allocates memory for the grid on the host (CPU).
    initializeGrid(h_grid, GRID_WIDTH, GRID_HEIGHT); //Initializes the grid with a random pattern.

    // Allocate memory for the grid on the device (GPU)
    unsigned char* d_grid;
    unsigned char* d_newGrid;
    hipMalloc(&d_grid, GRID_WIDTH * GRID_HEIGHT * sizeof(unsigned char)); //Allocates memory on the GPU for both the current grid
    hipMalloc(&d_newGrid, GRID_WIDTH * GRID_HEIGHT * sizeof(unsigned char)); //Copies the grid data from the host (CPU) to the GPU.
    // Copy the initial grid from host to device
    hipMemcpy(d_grid, h_grid, GRID_WIDTH * GRID_HEIGHT * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Set up CUDA execution configuration
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((GRID_WIDTH + BLOCK_SIZE - 1) / BLOCK_SIZE, (GRID_HEIGHT + BLOCK_SIZE - 1) / BLOCK_SIZE);

    bool quit = false;  // Flag to exit the main loop
    SDL_Event e;        // SDL event handler
    int frameCount = 0; // Counter for frames

    // Main loop that continues until the user closes the window.
    while (!quit) {
        // Handle events on queue
        while (SDL_PollEvent(&e) != 0) { //Handles SDL events, like closing the window.
            if (e.type == SDL_QUIT) {
                quit = true;  // User requests quit
            }
        }

        // Update the grid every UPDATE_FREQUENCY frames
        if (frameCount % UPDATE_FREQUENCY == 0) { //Checks if it's time to update the grid.
            // Launch CUDA kernel
            updateGrid << <gridSize, blockSize >> > (d_grid, d_newGrid, GRID_WIDTH, GRID_HEIGHT);
            // Swap grid pointers
            unsigned char* temp = d_grid;
            d_grid = d_newGrid;
            d_newGrid = temp;
            // Copy updated grid from device to host
            hipMemcpy(h_grid, d_grid, GRID_WIDTH * GRID_HEIGHT * sizeof(unsigned char), hipMemcpyDeviceToHost);
        }

        // Clear screen
        SDL_SetRenderDrawColor(renderer, 0, 0, 0, 255);  // Set color to black
        SDL_RenderClear(renderer);

        // Render grid
        SDL_SetRenderDrawColor(renderer, 255, 0, 0, 255);  // Set color to white
        for (int y = 0; y < GRID_HEIGHT; y++) {
            for (int x = 0; x < GRID_WIDTH; x++) {
                if (h_grid[y * GRID_WIDTH + x]) {  // If cell is alive
                    // Create a rectangle for the cell
                    SDL_Rect cellRect = { x * CELL_SIZE, y * CELL_SIZE, CELL_SIZE, CELL_SIZE };
                    SDL_RenderFillRect(renderer, &cellRect);  // Draw the cell
                }
            }
        }

        // Update screen
        SDL_RenderPresent(renderer);
        SDL_Delay(FRAME_DELAY);  // Add delay to control frame rate
        frameCount++;  // Increment frame counter
    }

    // Frees the GPU memory used for the grids.
    hipFree(d_grid);
    hipFree(d_newGrid);
    // Frees the memory used by the host grid.
    delete[] h_grid;

    // Destroy renderer
    SDL_DestroyRenderer(renderer);
    // Destroy window
    SDL_DestroyWindow(window);
    // Quit SDL subsystems
    SDL_Quit();

    return 0;  // Exit program
}
